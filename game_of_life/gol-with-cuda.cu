#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include <hip/hip_runtime.h>


extern "C" unsigned int *g_resultData;

// Current state of world.
extern "C" unsigned int *g_data;

// Current width of world.
extern "C" size_t g_worldWidth;

/// Current height of world.
extern "C" size_t g_worldHeight;

/// Current data length (product of width and height)
extern "C" size_t g_dataLength;

static inline void gol_initAllZeros( size_t worldWidth, size_t worldHeight )
{
    // calloc inits
    hipMallocManaged(&g_data, (g_dataLength+(2*worldWidth))*sizeof(unsigned int));
    hipMallocManaged(&g_resultData, (g_dataLength+(2*worldWidth))*sizeof(unsigned int));
    //set inits to 0
    for(size_t i = 0; i < g_dataLength; i++)
    {
        g_data[i+g_worldWidth] = 0;
        g_resultData[i+g_worldWidth] = 0;
    }
}

static inline void gol_initAllOnes( size_t worldWidth, size_t worldHeight )
{
    int i;

    hipMallocManaged(&g_data, (g_dataLength+(2*worldWidth))*sizeof(unsigned int));
    hipMallocManaged(&g_resultData, (g_dataLength+(2*worldWidth))*sizeof(unsigned int));

    // set all rows of world to true
    for( i = 0; i < g_dataLength; i++)
    {
        g_data[i+g_worldWidth] = 1;
        g_resultData[i+g_worldWidth]=0;
    }
    
    
}

static inline void gol_initOnesInMiddle( size_t worldWidth, size_t worldHeight )
{
    int i;
    
    hipMallocManaged(&g_data, (g_dataLength+(2*worldWidth))*sizeof(unsigned int));
    hipMallocManaged(&g_resultData, (g_dataLength+(2*worldWidth))*sizeof(unsigned int));
	
	
	for(i =0; i<10;i++){
		g_data[g_dataLength+127+i]=1;
	}
    
}

static inline void gol_initOnesAtCorners( size_t worldWidth, size_t worldHeight, int my_rank, int max_rank)
{
    hipMallocManaged(&g_data, (g_dataLength+(2*worldWidth))*sizeof(unsigned int));
    hipMallocManaged(&g_resultData, (g_dataLength+(2*worldWidth))*sizeof(unsigned int));

    if(my_rank==0){
		g_data[worldWidth] = 1; // upper left
    	g_data[(worldWidth*2)-1]=1; // upper right
	}
    if(my_rank == max_rank){
		
	    g_data[g_dataLength]=1; // lower left
	    g_data[g_dataLength + worldWidth-1]=1; // lower right
	}
    
    
}

static inline void gol_initSpinnerAtCorner( size_t worldWidth, size_t worldHeight, int my_rank)
{

    hipMallocManaged(&g_data, (g_dataLength+(2*worldWidth))*sizeof(unsigned int));
    hipMallocManaged(&g_resultData, (g_dataLength+(2*worldWidth))*sizeof(unsigned int));

	if(my_rank==0){
    	g_data[worldWidth] = 1; // upper left
    	g_data[1+worldWidth] = 1; // upper left +1
    	g_data[(worldWidth*2)-1]=1; // upper right
	}
    
}

static inline void gol_initMaster( unsigned int pattern, size_t worldWidth, size_t worldHeight, int myrank, int size)
{
	
    g_worldWidth = worldWidth;
    //g_worldHeight = worldHeight*size;
	g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;
	
	
	int cE, cudaDeviceCount;
	if((cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
			cE, cudaDeviceCount );
		exit(-1);
	}
	if((cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess ){
		printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
			myrank, (myrank % cudaDeviceCount), cE);
		exit(-1);
	}
	
    switch(pattern)
    {
    case 0:
    gol_initAllZeros( worldWidth, worldHeight );
    break;
    
    case 1:
    gol_initAllOnes( worldWidth, worldHeight );
    break;
    
    case 2:
    gol_initOnesInMiddle( worldWidth, worldHeight );
    break;
    
    case 3:
    gol_initOnesAtCorners( worldWidth, worldHeight, myrank, size-1);
    break;

    case 4:
    gol_initSpinnerAtCorner( worldWidth, worldHeight, myrank);
    break;

    default:
    printf("Pattern %u has not been implemented \n", pattern);
    exit(-1);
    }
}

extern "C" void initMaster( unsigned int pattern, size_t worldWidth, size_t worldHeight, int myrank, int size, int my_rank){
	gol_initMaster(pattern, worldWidth, worldHeight, myrank, size);
}

__device__ unsigned int gol_countAliveCells(const unsigned int* data,
                       size_t x0,
                       size_t x1,
                       size_t x2,
                       size_t y0,
                       size_t y1,
                       size_t y2)
{
    int counter = 0;
    // You write this function - it should return the number of alive cell for data[x1+y1]
    // There are 8 neighbors - see the assignment description for more details.
    
    //This increments counter based on the value of the 8 neighbors of the current element
    // if the neighbor is alive counter in incremented by 1
    counter+= (data[y0 + x0]+data[y0 + x1]+data[y0 + x2]);
    counter+= (data[y1 + x0]+data[y1 + x2]);
    counter+= (data[y2 + x0]+data[y2 + x1]+data[y2 + x2]);
    
    // the status of the node follow a tic is returns bellow according the specification
    // conditions
    if(data[x1+y1]==1){
        if(counter >=2 && counter <=3) return 1;
    }
    else
        if(counter==3) return 1;
    
    return 0;
}

__global__ void gol_kernel(const unsigned int * d_data, unsigned int worldWidth,
                           unsigned int worldHeight, unsigned int * d_resultData, int my_rank){
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    size_t dataLength = worldWidth * worldHeight;

    size_t y0,y1,y2;
    size_t x0,x1,x2;
    for(unsigned int i = index; i<worldHeight*worldWidth; i+=stride){
        
        x1 = index % worldWidth;
        y1 = index - x1;
        x0 = (x1 + worldWidth - 1) % worldWidth;
        x2 = (x1 + 1) % worldWidth;
        y0 = (y1 + dataLength - worldWidth) % dataLength;
        y2 = (y1 + worldWidth) % dataLength;
		
		unsigned int tmp = gol_countAliveCells(d_data, x0, x1, x2, y0, y1, y2);
        d_resultData[y1+x1] = tmp;
		
    }
}

extern "C" void kernelCall(int numBlocks, ushort threadsCount, const unsigned int *d_data, size_t worldWidth, size_t worldHeight, unsigned int** d_resultData, int my_rank){
	
	gol_kernel<<<numBlocks, threadsCount>>>(d_data, worldWidth, worldHeight, *d_resultData, my_rank);
	//cudaDeviceSynchronize();
}


